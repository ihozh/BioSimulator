#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
 
__global__ void addKernel( int* c, constint* a, const int* b )
{
         inti = threadIdx.x;
         c[i]= a[i] + b[i];
}
 
hipError_t CUDA_Add( const int* a, constint* b, int* out, int size )
{
         int*dev_a;
         int*dev_b;
         int*dev_c;
 
         //1、设置设备
         cudaError_tcudaStatus = hipSetDevice( 0 );
 
         switch( true )
         {
         default:
                   if( cudaStatus != hipSuccess )
                   {
                            fprintf(stderr, "调用hipSetDevice()函数失败！" );
                            returncudaStatus;
                   }
 
                   //2、分配显存空间
                   cudaStatus= hipMalloc( (void**)&dev_a, size * sizeof(int) );
                   if( cudaStatus != hipSuccess )
                   {
                            fprintf(stderr, "调用hipMalloc()函数初始化显卡中a数组时失败！" );
                            break;
                   }
 
                   cudaStatus= hipMalloc( (void**)&dev_b, size * sizeof(int) );
                   if( cudaStatus != hipSuccess )
                   {
                            fprintf(stderr, "调用hipMalloc()函数初始化显卡中b数组时失败！" );
                            break;
                   }
 
                   cudaStatus= hipMalloc( (void**)&dev_c, size * sizeof(int) );
                   if( cudaStatus != hipSuccess )
                   {
                            fprintf(stderr, "调用hipMalloc()函数初始化显卡中c数组时失败！" );
                            break;
                   }
 
                   //3、将宿主程序数据复制到显存中
                   cudaStatus= hipMemcpy( dev_a, a, size * sizeof( int ), hipMemcpyHostToDevice );
                   if( cudaStatus != hipSuccess )
                   {
                            fprintf( stderr, "调用hipMemcpy()函数初始化宿主程序数据a数组到显卡时失败！");
                            break;
                   }
                   cudaStatus= hipMemcpy( dev_b, b, size * sizeof( int ), hipMemcpyHostToDevice );
                   if( cudaStatus != hipSuccess )
                   {
                            fprintf(stderr, "调用hipMemcpy()函数初始化宿主程序数据b数组到显卡时失败！" );
                            break;
                   }
 
                   //4、执行程序，宿主程序等待显卡执行完毕
                   addKernel<<<1,size>>>( dev_c, dev_a, dev_b );
 
                   //5、查询内核初始化的时候是否出错
                   cudaStatus= hipGetLastError( );
                   if( cudaStatus != hipSuccess )
                   {
                            fprintf(stderr, "显卡执行程序时失败！" );
                            break;
                   }
 
                   //6、与内核同步等待执行完毕
                   cudaStatus= hipDeviceSynchronize( );
                   if( cudaStatus != hipSuccess )
                   {
                            fprintf(stderr, "在与内核同步的过程中发生问题！" );
                            break;
                   }
 
                   //7、获取数据
                   cudaStatus= hipMemcpy( out, dev_c, size * sizeof( int ), hipMemcpyDeviceToHost );
                   if( cudaStatus != hipSuccess )
                   {
                            fprintf(stderr, "在将结果数据从显卡复制到宿主程序中失败！" );
                            break;
                   }
         }
 
         hipFree(dev_c );
         hipFree(dev_a );
         hipFree(dev_b );
 
         returncudaStatus;
}
 
int main( int argc, char** argv )
{
         constint arraySize = 5;
         constint a[arraySize] = { 1, 2, 3, 4, 5 };
         constint b[arraySize] = { 10, 20, 30, 40, 50 };
         intc[arraySize] = { 0 };
 
         cudaError_tcudaStatus;
 
         cudaStatus= CUDA_Add( a, b, c, arraySize );
 
         printf("运算结果是：\nc数组[%d, %d, %d, %d, %d]\n",
                   c[0],c[1], c[2], c[3], c[4] );
 
         cudaStatus= hipDeviceReset( );
         if( cudaStatus != hipSuccess )
         {
                   fprintf(stderr, "调用hipDeviceReset()函数失败！" );
                   return1;
         }
 
         return0;
}
