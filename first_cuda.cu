#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE 1048576

int data[DATA_SIZE];

__global__ static void sumOfSquares(int *num, int* result)
{
    int sum = 0;
    int i;
    for(i = 0; i < DATA_SIZE; i++) {
        sum += num[i] * num[i];
    }

    *result = sum;
}
void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}
bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
              break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}

int main()
{
    if(!InitCUDA()) {
        return 0;
    }

    printf("CUDA initialized.\n");
    
    GenerateNumbers(data, DATA_SIZE);
    int* gpudata, *result;
    hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result, sizeof(int));
    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE,hipMemcpyHostToDevice);
    
    sumOfSquares<<<1, 1, 0>>>(gpudata, result);

    int sum;
    hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);

    printf("sum: %d\n", sum);

    return 0;
}
