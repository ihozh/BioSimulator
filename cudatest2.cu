#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cctype>
#include <cassert>
#include <cstdio>
#include <ctime>

#define DATA_SIZE 1048576
#define THREAD_NUM 256
#ifndef nullptr
#define nullptr 0
#endif

using namespace std;

void GenerateData( int* pData, size_t dataSize )// 产生数据
{
	assert( pData != nullptr );
	for ( size_t i = 0; i < dataSize; i++ )
	{
		srand( i + 3 );
		pData[i] = rand( ) % 100;
	}
}

////////////////////////在设备上运行的内核函数/////////////////////////////
__global__ static void Kernel_SquareSum( int* pIn, size_t* pDataSize,
										int* pOut, clock_t* pElapsed )
{
	// 开始计时
	clock_t startTime = clock( );

	for ( size_t i = 0; i < *pDataSize; ++i )
	{
		*pOut += pIn[i] * pIn[i];
	}

	*pElapsed = clock( ) - startTime;// 结束计时，返回至主程序
}

bool CUDA_SquareSum( int* pOut, clock_t* pElapsed,
					int* pIn, size_t dataSize )
{
	assert( pIn != nullptr );
	assert( pOut != nullptr );

	int* pDevIn = nullptr;
	int* pDevOut = nullptr;
	size_t* pDevDataSize = nullptr;
	clock_t* pDevElasped = nullptr;

	// 1、设置设备
	hipError_t cudaStatus = hipSetDevice( 0 );// 只要机器安装了英伟达显卡，那么会调用成功
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr, "调用hipSetDevice()函数失败！" );
		return false;
	}

	switch ( true )
	{
	default:
		// 2、分配显存空间
		cudaStatus = hipMalloc( (void**)&pDevIn, dataSize * sizeof( int ) );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "调用hipMalloc()函数初始化显卡中数组时失败！" );
			break;
		}

		cudaStatus = hipMalloc( (void**)&pDevOut, sizeof( int ) );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "调用hipMalloc()函数初始化显卡中返回值时失败！" );
			break;
		}

		cudaStatus = hipMalloc( (void**)&pDevDataSize, sizeof( size_t ) );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "调用hipMalloc()函数初始化显卡中数据大小时失败！" );
			break;
		}

		cudaStatus = hipMalloc( (void**)&pDevElasped, sizeof( clock_t ) );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "调用hipMalloc()函数初始化显卡中耗费用时变量失败！" );
			break;
		}

		// 3、将宿主程序数据复制到显存中
		cudaStatus = hipMemcpy( pDevIn, pIn, dataSize * sizeof( int ), hipMemcpyHostToDevice );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "调用hipMemcpy()函数初始化宿主程序数据数组到显卡时失败！" );
			break;
		}

		cudaStatus = hipMemcpy( pDevDataSize, &dataSize, sizeof( size_t ), hipMemcpyHostToDevice );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "调用hipMemcpy()函数初始化宿主程序数据大小到显卡时失败！" );
			break;
		}

		// 4、执行程序，宿主程序等待显卡执行完毕
		Kernel_SquareSum<<<1, 1>>>( pDevIn, pDevDataSize, pDevOut, pDevElasped );

		// 5、查询内核初始化的时候是否出错
		cudaStatus = hipGetLastError( );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "显卡执行程序时失败！" );
			break;
		}

		// 6、与内核同步等待执行完毕
		cudaStatus = hipDeviceSynchronize( );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "在与内核同步的过程中发生问题！" );
			break;
		}

		// 7、获取数据
		cudaStatus = hipMemcpy( pOut, pDevOut, sizeof( int ), hipMemcpyDeviceToHost );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "在将结果数据从显卡复制到宿主程序中失败！" );
			break;
		}

		cudaStatus = hipMemcpy( pElapsed, pDevElasped, sizeof( clock_t ), hipMemcpyDeviceToHost );
		if ( cudaStatus != hipSuccess )
		{
			fprintf( stderr, "在将耗费用时数据从显卡复制到宿主程序中失败！" );
			break;
		}

		hipFree( pDevIn );
		hipFree( pDevOut );
		hipFree( pDevDataSize );
		hipFree( pDevElasped );
		return true;
	}

	hipFree( pDevIn );
	hipFree( pDevOut );
	hipFree( pDevDataSize );
	hipFree( pDevElasped );
	return false;
}

int main( int argc, char** argv )// 函数的主入口
{
	int* pData = nullptr;
	int* pResult = nullptr;
	clock_t* pElapsed = nullptr;

	// 使用CUDA内存分配器分配host端
	hipError_t cudaStatus = hipHostMalloc( &pData, DATA_SIZE * sizeof( int ) );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr, "在主机中分配资源失败！" );
		return 1;
	}

	cudaStatus = hipHostMalloc( &pResult, sizeof( int ) );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr, "在主机中分配资源失败！" );
		return 1;
	}

	cudaStatus = hipHostMalloc( &pElapsed, sizeof( clock_t ) );
	if ( cudaStatus != hipSuccess )
	{
		fprintf( stderr, "在主机中分配资源失败！" );
		return 1;
	}

	GenerateData( pData, DATA_SIZE );// 通过随机数产生数据
	CUDA_SquareSum( pResult, pElapsed, pData, DATA_SIZE );// 执行平方和

	// 判断是否溢出
	char* pOverFlow = nullptr;
	if ( *pResult < 0 ) pOverFlow = "（溢出）";
	else pOverFlow = "";

	// 显示基准测试
	printf( "用CUDA计算平方和的结果是：%d%s\n耗费用时：%d\n",
		*pResult, pOverFlow, *pElapsed );

	hipDeviceProp_t prop;
	if ( hipGetDeviceProperties( &prop, 0 ) == hipSuccess )
	{
		clock_t actualTime = *pElapsed / clock_t( prop.clockRate );
		printf( "实际执行时间为：%dms\n", actualTime );
		printf( "带宽为：%.2fMB/s\n",
			float( DATA_SIZE * sizeof( int ) >> 20 ) * 1000.0f / float( actualTime ) );
		printf( "GPU设备型号：%s\n", prop.name );
	}

	hipHostFree( pData );
	hipHostFree( pResult );
	hipHostFree( pElapsed );


	return 0;
}
